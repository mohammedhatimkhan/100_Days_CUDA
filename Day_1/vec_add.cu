#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;


__global__ 
void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    float A[3] = {1.0, 2.0, 3.0};
    float B[3] = {1.0, 2.0, 3.0};
    float C[3] = {};

    float* Ap = A;
    float* Bp = B;
    float* Cp = C;

    int n = 3;
    int size = n * sizeof(float);

    float *A_d, *B_d, *C_d;
    
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    hipMemcpy(A_d, Ap, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, Bp, size, hipMemcpyHostToDevice);

    vecAddKernel<<<ceil(n/32.0), 32>>>(A_d, B_d, C_d, n);

    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

    cout << "[" ;
    for (int i=0; i < n; i++) {
        cout << C[i];
        if (i < n - 1){
            cout << ", ";
        }
    }
    cout << "]" << "\n";

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return 0;
}
